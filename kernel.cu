
#include <hip/hip_runtime.h>
#include <iostream>
#include <math.h>
#include <assert.h>
#include <vector>
#include <algorithm>
#include <time.h>

struct Monomial
{
    std::vector<int> degrees;
    int coefficient;
};

bool monomialEqual(Monomial const& a, Monomial const& b)
{
    return a.degrees == b.degrees && a.coefficient == b.coefficient;
}

struct Polynomial
{
    std::vector<Monomial> monomials;
};

/*
  Определяет, равны ли полиномы a и b.
*/
bool polynomialEqual(Polynomial const& a, Polynomial const& b)
{
    if (a.monomials.size() != b.monomials.size()) {
        return false;
    }
    for (size_t i = 0; i < a.monomials.size(); ++i) {
        if (!monomialEqual(a.monomials[i], b.monomials[i])) {
            return false;
        }
    }
    return true;
}

bool isMonomialLess(Monomial const& a, Monomial const& b)
{
    for (size_t i = 0; i < a.degrees.size(); ++i) {
        if (a.degrees[i] < b.degrees[i]) {
            return false;
        }
        else if (a.degrees[i] > b.degrees[i]) {
            return true;
        }
    }
    return false;
}

/*
  Определяет, отсортированы ли одночлены polynomial по убыванию.
  Для этого необходимо, чтобы одночлены должны быть приведены.
*/
bool isSorted(Polynomial const& polynomial)
{
    for (size_t i = 0; i < polynomial.monomials.size() - 1; ++i) {
        if (!isMonomialLess(polynomial.monomials[i], polynomial.monomials[i+1])) {
            return false;
        }
    }
    return true;
}

/*
  Возвращает старший одночлен. Так как все многочлены отсортированы,
  то старшим одночлленом всегда будет первый одночлен.
*/
Monomial getMajorMonomial(Polynomial const& polynomial)
{
    return polynomial.monomials[0];
}

/*
  Возвращает наименьшее общее частное monomial1 и monomial2.
  monomial1 и monomial2 должны быть нормализованы,
  то есть коэффициент при них должен быть равен 1.
*/
Monomial normalisedMonomialLeastCommonMultiple(
    Monomial const& monomial1, Monomial const& monomial2)
{
    assert(monomial1.coefficient == 1);
    assert(monomial2.coefficient == 1);
    Monomial lcm = monomial1;
    for (size_t i = 0; i < monomial1.degrees.size(); ++i) {
        lcm.degrees[i] = std::max(monomial1.degrees[i], monomial2.degrees[i]);
    }
    return lcm;
}

/*
  Возвращает результат деления divisible на divisor.
  divisible и divisor должны быть нормализованы,
  то есть коэффициент при них должен быть равен 1.
*/
Monomial normalisedMonomialDivide(Monomial const& divisible, Monomial const& divisor)
{
    assert(divisible.coefficient == 1);
    assert(divisor.coefficient == 1);
    auto res = divisible;
    for (size_t i = 0; i < divisible.degrees.size(); ++i) {
        res.degrees[i] -= divisor.degrees[i];
    }
    return res;
}

Polynomial multiplyByNormalisedMonomial(Polynomial const& polynomial, Monomial const& monomial)
{
    assert(monomial.coefficient == 1);
    auto res = polynomial;
    for (size_t monomialI = 0; monomialI < polynomial.monomials.size(); ++monomialI) {
        for (size_t i = 0; i < monomial.degrees.size(); ++i) {
            res.monomials[monomialI].degrees[i] += monomial.degrees[i];
        }
    }
    return res;
}

void sortPolynomial(Polynomial& polynomial) {
    std::sort(polynomial.monomials.begin(), polynomial.monomials.end(), isMonomialLess);
}

void addMonomial(Polynomial& polynomial, Monomial const& monomial) {
    for (auto& m : polynomial.monomials) {
        if (m.degrees == monomial.degrees) {
            m.coefficient += monomial.coefficient;
            return;
        }
    }
    polynomial.monomials.push_back(monomial);
}

Polynomial addPolynomials(Polynomial const& polynomial1, Polynomial const& polynomial2) {
    Polynomial result{ polynomial1.monomials };
    for (const auto& monomial : polynomial2.monomials) {
        addMonomial(result, monomial);
    }
    sortPolynomial(result);
    return result;
}

Polynomial generateRandomSortedPolynomial(size_t nVariables, size_t maxVariableDegree,
    size_t prime, size_t maxNMonomials)
{
    size_t nMonomials = rand() % maxNMonomials + 1;
    Polynomial result{ {} };
    for (size_t i = 0; i < nMonomials; ++i) {
        std::vector<int> degrees;
        for (size_t var = 0; var < nVariables; ++var) {
            int degree = rand() % (maxVariableDegree + 1);
            degrees.push_back(degree);
        }
        int coefficient = 1 + rand() % (prime-1);
        Monomial monomial{ degrees, coefficient };
        addMonomial(result, monomial);
    }
    sortPolynomial(result);
    return result;
}

void outputMonomial(Monomial const& monomial) {
    char letter = 'a';
    if (monomial.coefficient != 1) {
        std::cout << monomial.coefficient;
    }
    for (size_t i = 0; i < monomial.degrees.size(); ++i) {
        if (monomial.degrees[i]) {
            std::cout << letter;
            if (monomial.degrees[i] > 1) {
                std::cout << "^" << monomial.degrees[i];
            }
            if (i != monomial.degrees.size() - 1) {
                std::cout << " ";
            }
        }
        ++letter;
    }
}

void outputPolynomial(Polynomial const& polynomial)
{
    for (size_t i = 0; i < polynomial.monomials.size(); ++i) {
        outputMonomial(polynomial.monomials[i]);
        if (i != polynomial.monomials.size() - 1) {
            std::cout << " + ";
        }
    }
    std::cout << std::endl;
}

void testIsMonomialLess()
{
    auto monomial1 = Monomial{ std::vector<int>{2, 3}, 1 };
    auto monomial2 = Monomial{ std::vector<int>{2, 2}, 1 };
    assert(isMonomialLess(monomial1, monomial2));
    assert(!isMonomialLess(monomial2, monomial1));
    auto monomial3 = Monomial{ std::vector<int>{2, 3}, 1 };
    auto monomial4 = Monomial{ std::vector<int>{1, 5}, 1 };
    assert(isMonomialLess(monomial3, monomial4));
    assert(!isMonomialLess(monomial4, monomial3));
}

void testIsSorted()
{
    auto polynomial1 = Polynomial{ { Monomial{{1, 5}, 1}, Monomial{{2, 4}, 1} } };
    assert(!isSorted(polynomial1));
    auto polynomial2 = Polynomial{ { Monomial{{3, 2}, 1}, Monomial{{2, 4}, 1} } };
    assert(isSorted(polynomial2));
}

void testNormalisedMonomialDivide()
{
    auto divisible1 = Monomial{ std::vector<int>{2, 3}, 1 };
    auto divisor1 = Monomial{ std::vector<int>{0, 0}, 1 };
    auto expected1 = Monomial{ std::vector<int>{2, 3}, 1 };
    auto result1 = normalisedMonomialDivide(divisible1, divisor1);
    assert(monomialEqual(result1, expected1));
    auto divisor2 = Monomial{ std::vector<int>{2, 3}, 1 };
    auto expected2 = Monomial{ std::vector<int>{0, 0}, 1 };
    auto result2 = normalisedMonomialDivide(divisible1, divisor2);
    assert(monomialEqual(result2, expected2));
    auto divisor3 = Monomial{ std::vector<int>{1, 1}, 1 };
    auto expected3 = Monomial{ std::vector<int>{1, 2}, 1 };
    auto result3 = normalisedMonomialDivide(divisible1, divisor3);
    assert(monomialEqual(result3, expected3));
}

void testMultiplyByNormalisedMonomial()
{
    auto polynomial1 = Polynomial{ { Monomial{{1, 4}, 1} } };
    auto monomial1 = Monomial{ {1, 1}, 1 };
    auto expected1 = Polynomial{ { Monomial{{2, 5}, 1} } };
    auto result1 = multiplyByNormalisedMonomial(polynomial1, monomial1);
    assert(polynomialEqual(result1, expected1));
    auto polynomial2 = Polynomial{ { Monomial{{3, 4}, 1}, Monomial{{1, 2}, 1} } };
    auto expected2 = Polynomial{ { Monomial{{4, 5}, 1}, Monomial{{2, 3}, 1} } };
    auto result2 = multiplyByNormalisedMonomial(polynomial2, monomial1);
    assert(polynomialEqual(result2, expected2));
    auto polynomial3 = Polynomial{ { Monomial{{0, 0}, 1} } };
    auto monomial3 = Monomial{ {0, 0}, 1 };
    auto expected3 = Polynomial{ { Monomial{{0, 0}, 1} } };
    auto result3 = multiplyByNormalisedMonomial(polynomial3, monomial3);
    assert(polynomialEqual(result3, expected3));
    auto polynomial4 = Polynomial{ {} };
    auto monomial4 = Monomial{ {5}, 1 };
    auto expected4 = Polynomial{ {} };
    auto result4 = multiplyByNormalisedMonomial(polynomial4, monomial4);
    assert(polynomialEqual(result4, expected4));
}

void testGenerateRandomSortedPolynomial()
{
    for (size_t i = 0; i < 20; ++i)
    {
        auto poly = generateRandomSortedPolynomial(3, 5, 19, 5);
        outputPolynomial(poly);
    }
    for (size_t i = 0; i < 100; ++i)
    {
        auto poly = generateRandomSortedPolynomial(32, 50, 19, 50);
        assert(isSorted(poly));
    }
}

void testAll()
{
    testIsMonomialLess();
    testIsSorted();
    testNormalisedMonomialDivide();
    testMultiplyByNormalisedMonomial();
    testGenerateRandomSortedPolynomial();
}

int main(void)
{
    srand(time(nullptr));
    testAll();
    return 0;
}