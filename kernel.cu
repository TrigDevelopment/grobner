
#include <hip/hip_runtime.h>
#include <iostream>
#include <math.h>
#include <assert.h>
#include <vector>
#include <algorithm>

struct Monomial 
{
  std::vector<int> degrees;
  int coefficient;
};

bool monomialEqual(Monomial const & a, Monomial const & b)
{
  return a.degrees == b.degrees && a.coefficient == b.coefficient;
}

struct Polynomial 
{
  std::vector<Monomial> monomials;
};

Monomial getMajorMonomial(Polynomial const & polynomial)
{
  return polynomial.monomials[0];
}

/*
  ���������� ���������� ����� ������� monomial1 � monomial2.
  monomial1 � monomial2 ������ ���� �������������,
  �� ���� ����������� ��� ��� ������ ���� ����� 1.
*/
Monomial normalisedMonomialLeastCommonMultiple(
  Monomial const & monomial1, Monomial const & monomial2)
{
  assert(monomial1.coefficient == 1);
  assert(monomial2.coefficient == 1);
  Monomial lcm = monomial1;
  for (size_t i = 0; i < monomial1.degrees.size(); ++i) {
    lcm.degrees[i] = std::max(monomial1.degrees[i], monomial2.degrees[i]);
  }
  return lcm;
}

/*
  ���������� ��������� ������� divisible �� divisor. 
  divisible � divisor ������ ���� �������������,
  �� ���� ����������� ��� ��� ������ ���� ����� 1.
*/
Monomial normalisedMonomialDivide(Monomial const & divisible, Monomial const & divisor)
{
  assert(divisible.coefficient == 1);
  assert(divisor.coefficient == 1);
  auto res = divisible;
  for (size_t i = 0; i < divisible.degrees.size(); ++i) {
    res.degrees[i] -= divisor.degrees[i];
  }
  return res;
}

void testNormalisedMonomialDivide()
{
  auto divisible1 = Monomial{ std::vector<int>{2, 3}, 1 };
  auto divisor1 = Monomial{ std::vector<int>{0, 0}, 1 };
  auto expected1 = Monomial{ std::vector<int>{2, 3}, 1 };
  auto result1 = normalisedMonomialDivide(divisible1, divisor1);
  assert(monomialEqual(result1, expected1));
  auto divisor2 = Monomial{ std::vector<int>{2, 3}, 1 };
  auto expected2 = Monomial{ std::vector<int>{0, 0}, 1 };
  auto result2 = normalisedMonomialDivide(divisible1, divisor2);
  assert(monomialEqual(result2, expected2));
  auto divisor3 = Monomial{ std::vector<int>{1, 1}, 1 };
  auto expected3 = Monomial{ std::vector<int>{1, 2}, 1 };
  auto result3 = normalisedMonomialDivide(divisible1, divisor3);
  assert(monomialEqual(result3, expected3));
}

void testAll() 
{
  testNormalisedMonomialDivide();
}

int main(void)
{
  testAll();

  return 0;
}