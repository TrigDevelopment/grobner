
#include <hip/hip_runtime.h>
#include <iostream>
#include <math.h>
#include <assert.h>
#include <vector>
#include <algorithm>

struct Monomial 
{
  std::vector<int> degrees;
  int coefficient;
};

bool monomialEqual(Monomial const & a, Monomial const & b)
{
  return a.degrees == b.degrees && a.coefficient == b.coefficient;
}

struct Polynomial 
{
  std::vector<Monomial> monomials;
};

/*
  ����������, ����� �� �������� a � b.
*/
bool polynomialEqual(Polynomial const & a, Polynomial const & b)
{
  if (a.monomials.size() != b.monomials.size()) {
    return false;
  }
  for (size_t i = 0; i < a.monomials.size(); ++i) {
    if (!monomialEqual(a.monomials[i], b.monomials[i])) {
      return false;
    }
  }
  return true;
}

bool isMonomialLess(Monomial const & a, Monomial const & b)
{
  return false;
}

/*
  ����������, ������������� �� ��������� polynomial �� ��������.
  ��� ����� ����������, ����� ��������� ������ ���� ���������.
*/
bool isSorted(Polynomial const & polynomial)
{
  return false;
}

/*
  ���������� ������� ��������. ��� ��� ��� ���������� �������������,
  �� ������� ����������� ������ ����� ������ ��������.
*/
Monomial getMajorMonomial(Polynomial const & polynomial)
{
  return polynomial.monomials[0];
}

/*
  ���������� ���������� ����� ������� monomial1 � monomial2.
  monomial1 � monomial2 ������ ���� �������������,
  �� ���� ����������� ��� ��� ������ ���� ����� 1.
*/
Monomial normalisedMonomialLeastCommonMultiple(
  Monomial const & monomial1, Monomial const & monomial2)
{
  assert(monomial1.coefficient == 1);
  assert(monomial2.coefficient == 1);
  Monomial lcm = monomial1;
  for (size_t i = 0; i < monomial1.degrees.size(); ++i) {
    lcm.degrees[i] = std::max(monomial1.degrees[i], monomial2.degrees[i]);
  }
  return lcm;
}

/*
  ���������� ��������� ������� divisible �� divisor. 
  divisible � divisor ������ ���� �������������,
  �� ���� ����������� ��� ��� ������ ���� ����� 1.
*/
Monomial normalisedMonomialDivide(Monomial const & divisible, Monomial const & divisor)
{
  assert(divisible.coefficient == 1);
  assert(divisor.coefficient == 1);
  auto res = divisible;
  for (size_t i = 0; i < divisible.degrees.size(); ++i) {
    res.degrees[i] -= divisor.degrees[i];
  }
  return res;
}

Polynomial multiplyByNormalisedMonomial(Polynomial const & polynomial, Monomial const & monomial)
{
  assert(monomial.coefficient == 1);
  auto res = polynomial;
  for (size_t monomialI = 0; monomialI < polynomial.monomials.size(); ++monomialI) {
    for (size_t i = 0; i < monomial.degrees.size(); ++i) {
      res.monomials[monomialI].degrees[i] += monomial.degrees[i];
    }
  }
  return res;
}

Polynomial generateRandomSortedPolynomial(int nVariables, int maxVariableDegree,
  int maxPrime, int maxNMonomials) 
{
  return Polynomial{{}};
}

void outputPolynomial(Polynomial const & polynomial)
{

}

void testIsMonomialLess()
{
  
}

void testIsSorted()
{

}

void testNormalisedMonomialDivide()
{
  auto divisible1 = Monomial{ std::vector<int>{2, 3}, 1 };
  auto divisor1 = Monomial{ std::vector<int>{0, 0}, 1 };
  auto expected1 = Monomial{ std::vector<int>{2, 3}, 1 };
  auto result1 = normalisedMonomialDivide(divisible1, divisor1);
  assert(monomialEqual(result1, expected1));
  auto divisor2 = Monomial{ std::vector<int>{2, 3}, 1 };
  auto expected2 = Monomial{ std::vector<int>{0, 0}, 1 };
  auto result2 = normalisedMonomialDivide(divisible1, divisor2);
  assert(monomialEqual(result2, expected2));
  auto divisor3 = Monomial{ std::vector<int>{1, 1}, 1 };
  auto expected3 = Monomial{ std::vector<int>{1, 2}, 1 };
  auto result3 = normalisedMonomialDivide(divisible1, divisor3);
  assert(monomialEqual(result3, expected3));
}

void testMultiplyByNormalisedMonomial()
{
  auto polynomial1 = Polynomial{{ Monomial{{1, 4}, 1} }};
  auto monomial1 = Monomial{{1, 1}, 1};
  auto expected1 = Polynomial{{ Monomial{{2, 5}, 1} }};
  auto result1 = multiplyByNormalisedMonomial(polynomial1, monomial1);
  assert(polynomialEqual(result1, expected1));
  auto polynomial2 = Polynomial{{ Monomial{{3, 4}, 1}, Monomial{{1, 2}, 1} }};
  auto expected2 = Polynomial{{ Monomial{{4, 5}, 1}, Monomial{{2, 3}, 1} }};
  auto result2 = multiplyByNormalisedMonomial(polynomial2, monomial1);
  assert(polynomialEqual(result2, expected2));
  auto polynomial3 = Polynomial{{ Monomial{{0, 0}, 1} }};
  auto monomial3 = Monomial{{0, 0}, 1};
  auto expected3 = Polynomial{{ Monomial{{0, 0}, 1} }};
  auto result3 = multiplyByNormalisedMonomial(polynomial3, monomial3);
  assert(polynomialEqual(result3, expected3));
  auto polynomial4 = Polynomial{{}};
  auto monomial4 = Monomial{{5}, 1};
  auto expected4 = Polynomial{{}};
  auto result4 = multiplyByNormalisedMonomial(polynomial4, monomial4);
  assert(polynomialEqual(result4, expected4));
}

void testGenerateRandomSortedPolynomial()
{
  for (size_t i = 0; i < 10; ++i)
  {
    auto poly = generateRandomSortedPolynomial(3, 5, 20, 5);
    outputPolynomial(poly);
  }
  for (size_t i = 0; i < 100; ++i)
  {
    auto poly = generateRandomSortedPolynomial(32, 50, 20, 50);
    assert(isSorted(poly));
  }
}

void testAll() 
{
  testIsMonomialLess();
  testIsSorted();
  testNormalisedMonomialDivide();
  testMultiplyByNormalisedMonomial();
  testGenerateRandomSortedPolynomial();
}

int main(void)
{
  testAll();

  return 0;
}