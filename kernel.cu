
#include <hip/hip_runtime.h>
#include <iostream>
#include <math.h>
#include <assert.h>
#include <vector>
#include <algorithm>

struct Monomial 
{
  std::vector<int> degrees;
  int coefficient;
};

bool monomialEqual(Monomial const & a, Monomial const & b)
{
  return a.degrees == b.degrees && a.coefficient == b.coefficient;
}

struct Polynomial 
{
  std::vector<Monomial> monomials;
};

/*
  ����������, ����� �� �������� a � b.
*/
bool polynomialEqual(Polynomial const & a, Polynomial const & b)
{
  if (a.monomials.size() != b.monomials.size()) {
    return false;
  }
  for (size_t i = 0; i < a.monomials.size(); ++i) {
    if (!monomialEqual(a.monomials[i], b.monomials[i])) {
      return false;
    }
  }
  return true;
}

/*
  ���������� ������� ��������. ��� ��� ��� ���������� �������������,
  �� ������� ����������� ������ ����� ������ ��������.
*/
Monomial getMajorMonomial(Polynomial const & polynomial)
{
  return polynomial.monomials[0];
}

/*
  ���������� ���������� ����� ������� monomial1 � monomial2.
  monomial1 � monomial2 ������ ���� �������������,
  �� ���� ����������� ��� ��� ������ ���� ����� 1.
*/
Monomial normalisedMonomialLeastCommonMultiple(
  Monomial const & monomial1, Monomial const & monomial2)
{
  assert(monomial1.coefficient == 1);
  assert(monomial2.coefficient == 1);
  Monomial lcm = monomial1;
  for (size_t i = 0; i < monomial1.degrees.size(); ++i) {
    lcm.degrees[i] = std::max(monomial1.degrees[i], monomial2.degrees[i]);
  }
  return lcm;
}

/*
  ���������� ��������� ������� divisible �� divisor. 
  divisible � divisor ������ ���� �������������,
  �� ���� ����������� ��� ��� ������ ���� ����� 1.
*/
Monomial normalisedMonomialDivide(Monomial const & divisible, Monomial const & divisor)
{
  assert(divisible.coefficient == 1);
  assert(divisor.coefficient == 1);
  auto res = divisible;
  for (size_t i = 0; i < divisible.degrees.size(); ++i) {
    res.degrees[i] -= divisor.degrees[i];
  }
  return res;
}

void testNormalisedMonomialDivide()
{
  auto divisible1 = Monomial{ std::vector<int>{2, 3}, 1 };
  auto divisor1 = Monomial{ std::vector<int>{0, 0}, 1 };
  auto expected1 = Monomial{ std::vector<int>{2, 3}, 1 };
  auto result1 = normalisedMonomialDivide(divisible1, divisor1);
  assert(monomialEqual(result1, expected1));
  auto divisor2 = Monomial{ std::vector<int>{2, 3}, 1 };
  auto expected2 = Monomial{ std::vector<int>{0, 0}, 1 };
  auto result2 = normalisedMonomialDivide(divisible1, divisor2);
  assert(monomialEqual(result2, expected2));
  auto divisor3 = Monomial{ std::vector<int>{1, 1}, 1 };
  auto expected3 = Monomial{ std::vector<int>{1, 2}, 1 };
  auto result3 = normalisedMonomialDivide(divisible1, divisor3);
  assert(monomialEqual(result3, expected3));
}

Polynomial multiplyByNormalisedMonomial(Polynomial const & polynomial, Monomial const & monomial)
{
  assert(monomial.coefficient == 1);
  auto res = polynomial;
  for (size_t monomialI = 0; monomialI < polynomial.monomials.size(); ++monomialI) {
    for (size_t i = 0; i < monomial.degrees.size(); ++i) {
      res.monomials[monomialI].degrees[i] += monomial.degrees[i];
    }
  }
  return res;
}

void testMultiplyByNormalisedMonomial()
{
  auto polynomial1 = Polynomial{{ Monomial{{1, 4}, 1} }};
  auto monomial1 = Monomial{{1, 1}, 1};
  auto expected1 = Polynomial{{ Monomial{{2, 5}, 1} }};
  auto result1 = multiplyByNormalisedMonomial(polynomial1, monomial1);
  assert(polynomialEqual(result1, expected1));
  auto polynomial2 = Polynomial{{ Monomial{{3, 4}, 1}, Monomial{{1, 2}, 1} }};
  auto expected2 = Polynomial{{ Monomial{{4, 5}, 1}, Monomial{{2, 3}, 1} }};
  auto result2 = multiplyByNormalisedMonomial(polynomial2, monomial1);
  assert(polynomialEqual(result2, expected2));
  auto polynomial3 = Polynomial{{ Monomial{{0, 0}, 1} }};
  auto monomial3 = Monomial{{0, 0}, 1};
  auto expected3 = Polynomial{{ Monomial{{0, 0}, 1} }};
  auto result3 = multiplyByNormalisedMonomial(polynomial3, monomial3);
  assert(polynomialEqual(result3, expected3));
  auto polynomial4 = Polynomial{{}};
  auto monomial4 = Monomial{{5}, 1};
  auto expected4 = Polynomial{{}};
  auto result4 = multiplyByNormalisedMonomial(polynomial4, monomial4);
  assert(polynomialEqual(result4, expected4));
}

void testAll() 
{
  testNormalisedMonomialDivide();
  testMultiplyByNormalisedMonomial();
}

int main(void)
{
  testAll();

  return 0;
}