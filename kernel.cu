
#include <hip/hip_runtime.h>
#include <iostream>
#include <math.h>
#include <assert.h>
#include <vector>
#include <algorithm>
#include <time.h>

struct Monomial
{
    std::vector<int> degrees;
    int coefficient;
};

bool monomialEqual(Monomial const& a, Monomial const& b)
{
    return a.degrees == b.degrees && a.coefficient == b.coefficient;
}

struct Polynomial
{
    std::vector<Monomial> monomials;
};

struct PolynomialBasis
{
  std::vector<Polynomial> polynomials;
};

void mod(int& number, int prime) {
    number %= prime;
    if (number < 0) {
        number += prime;
    }
}

/* Алгоритм Евклида*/
int gcdex(int a, int b, int& x, int& y) {
    if (b == 0) {
        x = 1;
        y = 0;
        return a;
    }
    int x1, y1;
    int d1 = gcdex(b, a % b, x1, y1);
    x = y1;
    y = x1 - (a / b) * y1;
    return d1;
}

int getInversedElement(int a, int N) {
    int x, y;
    gcdex(a, N, x, y);
    return x;
}

/*
  Определяет, равны ли полиномы a и b.
*/
bool polynomialEqual(Polynomial const& a, Polynomial const& b)
{
    if (a.monomials.size() != b.monomials.size()) {
        return false;
    }
    for (size_t i = 0; i < a.monomials.size(); ++i) {
        if (!monomialEqual(a.monomials[i], b.monomials[i])) {
            return false;
        }
    }
    return true;
}

bool isMonomialGreater(Monomial const& a, Monomial const& b)
{
    assert(a.degrees.size() == b.degrees.size());
    for (size_t i = 0; i < a.degrees.size(); ++i) {
        if (a.degrees[i] > b.degrees[i]) {
            return true;
        }
        else if (a.degrees[i] < b.degrees[i]) {
            return false;
        }
    }
    return false;
}

/*
  Определяет, отсортированы ли одночлены polynomial по убыванию.
  Для этого необходимо, чтобы одночлены должны быть приведены.
*/
bool isSorted(Polynomial const& polynomial)
{
    for (size_t i = 0; i + 1 < polynomial.monomials.size(); ++i) {
        if (!isMonomialGreater(polynomial.monomials[i], polynomial.monomials[i + 1])) {
            return false;
        }
    }
    return true;
}

/*
  Возвращает старший одночлен. Так как все многочлены отсортированы по убыванию,
  то старшим одночлленом всегда будет первый одночлен.
*/
Monomial getMajorMonomial(Polynomial const& polynomial)
{
  assert(polynomial.monomials.size() > 0);
  return polynomial.monomials[0];
}

bool isPolynomialGreater(Polynomial const& a, Polynomial const& b)
{
    auto majorA = getMajorMonomial(a);
    auto majorB = getMajorMonomial(b);
    return isMonomialGreater(majorA, majorB);
}

bool isSorted(PolynomialBasis const& basis)
{
    for (size_t i = 0; i + 1 < basis.polynomials.size(); ++i) {
        if (!isPolynomialGreater(basis.polynomials[i], basis.polynomials[i + 1])) {
            return false;
        }
    }
    return true;
}

/*
  Возвращает наименьшее общее частное monomial1 и monomial2.
  monomial1 и monomial2 должны быть нормализованы,
  то есть коэффициент при них должен быть равен 1.
*/
Monomial normalisedMonomialLeastCommonMultiple(
    Monomial const& monomial1, Monomial const& monomial2)
{
    assert(monomial1.coefficient == 1);
    assert(monomial2.coefficient == 1);
    Monomial lcm = monomial1;
    for (size_t i = 0; i < monomial1.degrees.size(); ++i) {
        lcm.degrees[i] = std::max(monomial1.degrees[i], monomial2.degrees[i]);
    }
    return lcm;
}

/*
  Возвращает результат деления divisible на divisor.
  divisor должен быть нормализован,
  то есть коэффициент при этом одночлене должен быть равен 1.
*/
Monomial dividedByNormalisedMonomial(Monomial const& divisible, Monomial const& divisor)
{
    assert(divisor.coefficient == 1);
    auto res = divisible;
    for (size_t i = 0; i < divisible.degrees.size(); ++i) {
        res.degrees[i] -= divisor.degrees[i];
    }
    return res;
}

Polynomial multiplyByNormalisedMonomial(Polynomial const& polynomial, Monomial const& monomial)
{
    assert(monomial.coefficient == 1);
    auto res = polynomial;
    for (size_t monomialI = 0; monomialI < polynomial.monomials.size(); ++monomialI) {
        for (size_t i = 0; i < monomial.degrees.size(); ++i) {
            res.monomials[monomialI].degrees[i] += monomial.degrees[i];
        }
    }
    return res;
}

void sortPolynomial(Polynomial& polynomial) 
{
    std::sort(polynomial.monomials.begin(), polynomial.monomials.end(), isMonomialGreater);
}

void sortPolynomialBasis(PolynomialBasis& basis) 
{
    std::sort(basis.polynomials.begin(), basis.polynomials.end(), isPolynomialGreater);
}

void addMonomial(Polynomial& polynomial, Monomial const& monomial, int prime) {
    for (size_t i = 0; i < polynomial.monomials.size(); ++i) {
        if (polynomial.monomials[i].degrees == monomial.degrees) {
            polynomial.monomials[i].coefficient += monomial.coefficient;
            mod(polynomial.monomials[i].coefficient, prime);
            if (!polynomial.monomials[i].coefficient) {
                polynomial.monomials.erase(polynomial.monomials.begin() + i);
            }
            return;
        }
    }
    polynomial.monomials.push_back(monomial);
}

/*
  Возвращает результат умножения polynomial на -1.
*/
Polynomial negate(Polynomial const& polynomial, int prime)
{
    auto current = polynomial;
    for (auto& monomial : current.monomials) {
        monomial.coefficient = prime - monomial.coefficient;
    }
    return current;
}

Polynomial addPolynomials(Polynomial const& polynomial1, Polynomial const& polynomial2, int prime) {
    Polynomial result{ polynomial1.monomials };
    for (const auto& monomial : polynomial2.monomials) {
        addMonomial(result, monomial, prime);
    }
    sortPolynomial(result);
    return result;
}

Polynomial generateRandomSortedPolynomial(size_t nVariables, size_t maxVariableDegree,
    size_t prime, size_t maxNMonomials)
{
    size_t nMonomials = rand() % (maxNMonomials + 1);
    Polynomial result{ {} };
    for (size_t i = 0; i < nMonomials; ++i) {
        std::vector<int> degrees;
        for (size_t var = 0; var < nVariables; ++var) {
            int degree = rand() % (maxVariableDegree + 1);
            degrees.push_back(degree);
        }
        int coefficient = 1 + rand() % (prime - 1);
        Monomial monomial{ degrees, coefficient };
        addMonomial(result, monomial, prime);
    }
    sortPolynomial(result);
    return result;
}

void outputMonomial(Monomial const& monomial) {
    char letter = 'a';
    if (monomial.coefficient != 1) {
        std::cout << monomial.coefficient;
    }
    for (size_t i = 0; i < monomial.degrees.size(); ++i) {
        if (monomial.degrees[i]) {
            std::cout << letter;
            if (monomial.degrees[i] > 1) {
                std::cout << "^" << monomial.degrees[i];
            }
            if (i != monomial.degrees.size() - 1) {
                std::cout << " ";
            }
        }
        ++letter;
    }
}

void outputPolynomial(Polynomial const& polynomial)
{
    for (size_t i = 0; i < polynomial.monomials.size(); ++i) {
        outputMonomial(polynomial.monomials[i]);
        if (i != polynomial.monomials.size() - 1) {
            std::cout << " + ";
        }
    }
    std::cout << std::endl;
}

void subtract(Polynomial& polynomial1, Polynomial const& polynomial2, int prime)
{
    polynomial1 = addPolynomials(polynomial1, negate(polynomial2, prime), prime);
}

Polynomial multipliedByMonomial(Polynomial const& polynomial, Monomial const& monomial, int prime)
{
    auto res = polynomial;
    for (size_t monomialI = 0; monomialI < polynomial.monomials.size(); ++monomialI) {
        for (size_t i = 0; i < monomial.degrees.size(); ++i) {
            res.monomials[monomialI].degrees[i] += monomial.degrees[i];
        }
        res.monomials[monomialI].coefficient *= monomial.coefficient;
        mod(res.monomials[monomialI].coefficient, prime);
        if (!res.monomials[monomialI].coefficient) {
            res.monomials.erase(res.monomials.begin() + monomialI);
        }
    }
    return res;
}

bool isMonomialDivide(Monomial const& monomial1, Monomial const& monomial2)
{
    assert(monomial1.degrees.size() == monomial2.degrees.size());
    for (size_t i = 0; i < monomial1.degrees.size(); ++i) {
        if (monomial1.degrees[i] > monomial2.degrees[i]) {
            return false;
        }
    }
    return true;
}

void normalise(Polynomial& polynomial, int prime)
{
    assert(isSorted(polynomial));
    if (polynomial.monomials.size() > 0) {
        auto const major = getMajorMonomial(polynomial);
        int coefficient = getInversedElement(major.coefficient, prime);
        for (auto& monomial : polynomial.monomials) {
            monomial.coefficient *= coefficient;
            mod(monomial.coefficient, prime);
        }
    }
}

Polynomial getReducedPolynomial(Polynomial const& polynomial,
    PolynomialBasis const& basis, int prime)
{
    assert(isSorted(basis));
    auto current = polynomial;
    for (size_t basisPolynomialI = 0;
        basisPolynomialI < basis.polynomials.size();
        ++basisPolynomialI) {
        auto basisPolynomial = basis.polynomials[basisPolynomialI];
        auto major = getMajorMonomial(basisPolynomial);
        for (size_t monomialI = 0; monomialI < current.monomials.size(); ++monomialI) {
            auto monomial = current.monomials[monomialI];
            if (isMonomialDivide(major, monomial)) {
                auto multiplier = dividedByNormalisedMonomial(monomial, major);
                auto multipliedBasisPolynomial =
                    multipliedByMonomial(basisPolynomial, multiplier, prime);
                subtract(current, multipliedBasisPolynomial, prime);
                monomialI = 0;
            }
        }
    }
    normalise(current, prime);
    return current;
}

void testNegate()
{
    auto poly1 = Polynomial{ {} };
    assert(polynomialEqual(negate(poly1, 7), poly1));
    auto poly2 = Polynomial{ { {{2, 3}, 1}, {{2, 2}, 4} } };
    auto poly3 = Polynomial{ { {{2, 3}, 6}, {{2, 2}, 3} } };
    assert(polynomialEqual(negate(poly2, 7), poly3));
}

void testIsMonomialLess()
{
    auto monomial1 = Monomial{ {2, 3}, 1 };
    auto monomial2 = Monomial{ {2, 2}, 1 };
    assert(isMonomialGreater(monomial1, monomial2));
    assert(!isMonomialGreater(monomial2, monomial1));
    auto monomial3 = Monomial{ {2, 3}, 1 };
    auto monomial4 = Monomial{ {1, 5}, 1 };
    assert(isMonomialGreater(monomial3, monomial4));
    assert(!isMonomialGreater(monomial4, monomial3));
}

void testIsSorted()
{
    auto polynomial1 = Polynomial{ { Monomial{{1, 5}, 1}, Monomial{{2, 4}, 2} } };
    assert(!isSorted(polynomial1));
    auto polynomial2 = Polynomial{ { Monomial{{3, 2}, 2}, Monomial{{2, 4}, 1} } };
    assert(isSorted(polynomial2));
}

void testNormalisedMonomialDivide()
{
    auto divisible1 = Monomial{ {2, 3}, 2 };
    auto divisor1 = Monomial{ {0, 0}, 1 };
    auto expected1 = Monomial{ {2, 3}, 2 };
    auto result1 = dividedByNormalisedMonomial(divisible1, divisor1);

    assert(monomialEqual(result1, expected1));
    auto divisor2 = Monomial{ {2, 3}, 1 };
    auto expected2 = Monomial{ {0, 0}, 2 };
    auto result2 = dividedByNormalisedMonomial(divisible1, divisor2);
    assert(monomialEqual(result2, expected2));

    auto divisor3 = Monomial{ {1, 1}, 1 };
    auto expected3 = Monomial{ {1, 2}, 2 };
    auto result3 = dividedByNormalisedMonomial(divisible1, divisor3);
    assert(monomialEqual(result3, expected3));
}

void testMultiplyByNormalisedMonomial()
{
    auto polynomial1 = Polynomial{ { Monomial{{1, 4}, 1} } };
    auto monomial1 = Monomial{ {1, 1}, 1 };
    auto expected1 = Polynomial{ { Monomial{{2, 5}, 1} } };
    auto result1 = multiplyByNormalisedMonomial(polynomial1, monomial1);
    assert(polynomialEqual(result1, expected1));
    auto polynomial2 = Polynomial{ { Monomial{{3, 4}, 1}, Monomial{{1, 2}, 1} } };
    auto expected2 = Polynomial{ { Monomial{{4, 5}, 1}, Monomial{{2, 3}, 1} } };
    auto result2 = multiplyByNormalisedMonomial(polynomial2, monomial1);
    assert(polynomialEqual(result2, expected2));
    auto polynomial3 = Polynomial{ { Monomial{{0, 0}, 1} } };
    auto monomial3 = Monomial{ {0, 0}, 1 };
    auto expected3 = Polynomial{ { Monomial{{0, 0}, 1} } };
    auto result3 = multiplyByNormalisedMonomial(polynomial3, monomial3);
    assert(polynomialEqual(result3, expected3));
    auto polynomial4 = Polynomial{ {} };
    auto monomial4 = Monomial{ {5}, 1 };
    auto expected4 = Polynomial{ {} };
    auto result4 = multiplyByNormalisedMonomial(polynomial4, monomial4);
    assert(polynomialEqual(result4, expected4));
}

void testGenerateRandomSortedPolynomial()
{
    for (size_t i = 0; i < 20; ++i)
    {
        auto poly = generateRandomSortedPolynomial(3, 5, 19, 5);
        outputPolynomial(poly);
    }
    for (size_t i = 0; i < 100; ++i)
    {
        auto poly = generateRandomSortedPolynomial(32, 50, 19, 50);
        assert(isSorted(poly));
    }
}

void testSubtract()
{
    auto poly1 = Polynomial{{}};
    subtract(poly1, {{}}, 7);
    assert(polynomialEqual(poly1, {{}}));

    auto poly2 = Polynomial{{}};
    subtract(poly2, {{ {{2, 3}, 5} }}, 7);
    assert(polynomialEqual(poly2, {{ {{2, 3}, 2} }}));

    auto poly3 = Polynomial{{ {{2, 3}, 5} }};
    subtract(poly3, {{}}, 7);
    assert(polynomialEqual(poly3, {{ {{2, 3}, 5} }}));

    auto poly4 = Polynomial{{ {{2, 3}, 5} }};
    subtract(poly4, {{ {{2, 3}, 2} }}, 7);
    assert(polynomialEqual(poly4, {{ {{2, 3}, 3} }}));

    auto poly5 = Polynomial{{ {{2, 3}, 5} }};
    subtract(poly5, {{ {{2, 3}, 5} }}, 7);
    assert(polynomialEqual(poly5, {{}}));

    auto poly6 = Polynomial{{ {{2, 3}, 2} }};
    subtract(poly6, {{ {{2, 3}, 3} }}, 7);
    assert(polynomialEqual(poly6, {{ {{2, 3}, 6} }}));

    auto poly7 = Polynomial{{ {{3, 3}, 1} }};
    subtract(poly7, {{ {{2, 2}, 1} }}, 5);
    assert(polynomialEqual(poly7, {{ {{3, 3}, 1}, {{2, 2}, 4} }}));

    auto poly8 = Polynomial{{ {{2, 2}, 1} }};
    subtract(poly8, {{ {{3, 3}, 1} }}, 5);
    assert(polynomialEqual(poly8, {{ {{3, 3}, 4}, {{2, 2}, 1} }}));

    auto poly9 = Polynomial{{ {{4, 4}, 2}, {{2, 2}, 3} }};
    subtract(poly9, {{ {{3, 3}, 4}, {{1, 1}, 3} }}, 5);
    assert(polynomialEqual(poly9, {{ {{4, 4}, 2}, {{3, 3}, 1}, {{2, 2}, 3}, {{1, 1}, 2} }}));

    auto poly10 = Polynomial{{ {{1, 2}, 1} }};
    subtract(poly10, {{ {{2, 1}, 1} }}, 5);
    assert(polynomialEqual(poly10, {{ {{2, 1}, 4}, {{1, 2}, 1} }}));
}

void testMultipliedByMonomial()
{
    auto poly0 = Polynomial{ { {{0, 0}, 1} } };
    auto monomial0 = Monomial{ {0, 0}, 1 };
    auto expected0 = Polynomial{ { {{0, 0}, 1} } };
    auto result0 = multipliedByMonomial(poly0, monomial0, 5);
    assert(polynomialEqual(expected0, result0));

    auto poly1 = Polynomial{ { {{1, 4}, 1}} };
    auto monomial1 = Monomial{ {1, 1}, 1 };
    auto expected1 = Polynomial{ { {{2, 5}, 1} } };
    auto result1 = multipliedByMonomial(poly1, monomial1, 5);
    assert(polynomialEqual(expected1, result1));

    auto poly2 = Polynomial{ { {{3, 4}, 1}, {{1, 2}, 1} } };
    auto monomial2 = Monomial{ {1, 1}, 1 };
    auto expected2 = Polynomial{ { {{4, 5}, 1}, {{2, 3}, 1} } };
    auto result2 = multipliedByMonomial(poly2, monomial2, 5);
    assert(polynomialEqual(expected2, result2));

    auto poly3 = Polynomial{ { {{1, 1}, 1} } };
    auto monomial3 = Monomial{ {0, 0}, 2 };
    auto expected3 = Polynomial{ { {{1, 1}, 2} } };
    auto result3 = multipliedByMonomial(poly3, monomial3, 5);
    assert(polynomialEqual(expected3, result3));
}

void testGetReducedPolynomial()
{
    auto poly0 = Polynomial{{}};
    auto poly1 = Polynomial{{ {{0}, 1} }};
    auto result1 = getReducedPolynomial(poly1, { { poly1 } }, 7);
    assert(polynomialEqual(result1, poly0));

    auto poly2 = Polynomial{{ {{1}, 1} }};
    auto result2 = getReducedPolynomial(poly2, { { poly1 } }, 7);
    assert(polynomialEqual(result2, poly0));

    auto poly3 = Polynomial{{ {{2}, 1}}};
    auto poly4 = Polynomial{{ {{5}, 1} }};
    auto result3 = getReducedPolynomial(poly4, { { poly3 } }, 7);
    assert(polynomialEqual(result3, poly0));

    auto result4 = getReducedPolynomial(poly3, { { poly4 } }, 7);
    assert(polynomialEqual(result4, poly3));

    auto poly5 = Polynomial{ { {{1, 0}, 1}, {{0, 1}, 1} } };
    auto poly6 = Polynomial{ { {{1, 0}, 1} }};
    auto poly7 = Polynomial{{ {{0, 1}, 1} }};
    auto result5 = getReducedPolynomial(poly6, { { poly5 } }, 7);
    assert(polynomialEqual(result5, poly7));

    auto poly8 = Polynomial{ { {{0, 1, 0}, 1}, {{0, 0, 1}, 1} } };
    auto poly9 = Polynomial{ { {{1, 0, 0}, 1}, {{0, 1, 0}, 1} } };
    auto poly10 = Polynomial{ { {{1, 0, 0}, 1} } };
    auto poly11 = Polynomial{ { {{0, 0, 1}, 1} } };
    auto basis0 = PolynomialBasis{ { poly8, poly9 } };
    sortPolynomialBasis(basis0);
    auto result6 = getReducedPolynomial(poly10, basis0, 7);
    assert(polynomialEqual(result6, poly11));

    auto result7 = getReducedPolynomial(poly8, basis0, 7);
    assert(polynomialEqual(result7, {{}}));

    auto poly12 = Polynomial{ { {{1, 0}, 3} } };
    auto poly13 = Polynomial{ { {{0, 1}, 1} } };
    auto result8 = getReducedPolynomial(poly12, { { poly5 } }, 7);
    assert(polynomialEqual(result8, poly13));
}

void testAll()
{
    testNegate();
    testIsMonomialLess();
    testIsSorted();
    testNormalisedMonomialDivide();
    testMultiplyByNormalisedMonomial();
    testGenerateRandomSortedPolynomial();
    testSubtract();
    testMultipliedByMonomial();
    testGetReducedPolynomial();
}

int main(void)
{
    srand(time(nullptr));
    testAll();
    return 0;
}