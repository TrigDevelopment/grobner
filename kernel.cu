#include "hip/hip_runtime.h"
#include <iostream>
#include <math.h>
#include <assert.h>
#include <vector>
#include <algorithm>
#include <time.h>
#include <string>

bool isPositiveInteger(const std::string& s)
{
    return !s.empty() &&
        (std::count_if(s.begin(), s.end(), std::isdigit) == s.size());
}

std::vector<std::string> split(const std::string& input, const std::string& delimiter) {
    std::vector<std::string> tokens;
    size_t prev = 0, pos = 0;
    do {
        pos = input.find(delimiter, prev);
        if (pos == std::string::npos) {
            pos = input.length();
        }
        std::string token = input.substr(prev, pos - prev);
        if (!token.empty()) {
            tokens.push_back(token);
        }
        prev = pos + delimiter.length();
    } while (pos < input.length() && prev < input.length());
    return tokens;
}

struct Monomial
{
    std::vector<int> degrees;
    int coefficient;

    Monomial() : coefficient(0) {}

    Monomial(std::vector<int> degrees, int coefficient) : degrees(degrees), coefficient(coefficient) {}

    Monomial(const std::string& input, size_t nVariables) {
        auto variables = split(input, "*");
        if (isPositiveInteger(variables.at(0))) {
            coefficient = atoi(variables.at(0).c_str());
            variables.erase(variables.begin());
        }
        else {
            coefficient = 1;
        }
        char current_letter = 'a';
        for (const auto& variable : variables) {
            auto tokens = split(variable, "^");
            while (tokens.at(0)[0] != current_letter) {
                degrees.push_back(0);
                ++current_letter;
            }
            degrees.push_back(atoi(tokens.at(1).c_str()));
            ++current_letter;
        }
        while (degrees.size() < nVariables) {
            degrees.push_back(0);
        }
    }
};

bool operator==(const Monomial& a, const Monomial& b)
{
    return a.degrees == b.degrees && a.coefficient == b.coefficient;
}

struct Polynomial
{
    std::vector<Monomial> monomials;

    Polynomial() {}
    Polynomial(std::vector<Monomial> monomials) : monomials(monomials) {}
    Polynomial(const std::string& input, size_t nVariables) {
        auto monomialTokens = split(input, " + ");
        for (const auto& token : monomialTokens) {
            monomials.push_back(Monomial(token, nVariables));
        }
    }
};

struct PolynomialBasis
{
    std::vector<Polynomial> polynomials;
};

void mod(int& number, int prime) {
    number %= prime;
    if (number < 0) {
        number += prime;
    }
}

/* Алгоритм Евклида*/
int gcdex(int a, int b, int& x, int& y) {
    if (b == 0) {
        x = 1;
        y = 0;
        return a;
    }
    int x1, y1;
    int d1 = gcdex(b, a % b, x1, y1);
    x = y1;
    y = x1 - (a / b) * y1;
    return d1;
}

int getInverseElement(int a, int N) {
    int x, y;
    gcdex(a, N, x, y);
    return x;
}

/*
  Определяет, равны ли полиномы a и b.
*/
bool polynomialEqual(Polynomial const& a, Polynomial const& b)
{
    if (a.monomials.size() != b.monomials.size()) {
        return false;
    }
    for (size_t i = 0; i < a.monomials.size(); ++i) {
        if (!(a.monomials[i] == b.monomials[i])) {
            return false;
        }
    }
    return true;
}

bool basisEqual(PolynomialBasis const& a, PolynomialBasis const& b)
{
    if (a.polynomials.size() != b.polynomials.size()) {
        return false;
    }
    for (size_t i = 0; i < a.polynomials.size(); ++i) {
        if (!polynomialEqual(a.polynomials[i], b.polynomials[i])) {
            return false;
        }
    }
    return true;
}

bool isMonomialGreater(Monomial const& a, Monomial const& b)
{
    assert(a.degrees.size() == b.degrees.size());
    for (size_t i = 0; i < a.degrees.size(); ++i) {
        if (a.degrees[i] > b.degrees[i]) {
            return true;
        }
        else if (a.degrees[i] < b.degrees[i]) {
            return false;
        }
    }
    return false;
}

/*
  Определяет, отсортированы ли одночлены polynomial по убыванию.
  Для этого необходимо, чтобы одночлены должны быть приведены.
*/
bool isSorted(Polynomial const& polynomial)
{
    for (size_t i = 0; i + 1 < polynomial.monomials.size(); ++i) {
        if (!isMonomialGreater(polynomial.monomials[i], polynomial.monomials[i + 1])) {
            return false;
        }
    }
    return true;
}

/*
  Возвращает старший одночлен. Так как все многочлены отсортированы по убыванию,
  то старшим одночлленом всегда будет первый одночлен.
*/
Monomial getMajorMonomial(Polynomial const& polynomial)
{
    assert(polynomial.monomials.size() > 0);
    return polynomial.monomials[0];
}

bool isPolynomialGreater(Polynomial const& a, Polynomial const& b)
{
    assert(isSorted(a));
    assert(isSorted(b));
    for (size_t i = 0; i < a.monomials.size(); ++i) {
        if (isMonomialGreater(a.monomials[i], b.monomials[i])) {
            return true;
        }
        else if (isMonomialGreater(b.monomials[i], a.monomials[i])) {
            return false;
        }
    }
    return false;
}

bool isSorted(PolynomialBasis const& basis)
{
    for (size_t i = 0; i + 1 < basis.polynomials.size(); ++i) {
        auto current = basis.polynomials[i];
        auto next = basis.polynomials[i + 1];
        if (!isPolynomialGreater(current, next) &&
            !polynomialEqual(current, next)) {
            return false;
        }
    }
    return true;
}

/*
  Возвращает наименьшее общее частное monomial1 и monomial2.
  monomial1 и monomial2 должны быть нормализованы,
  то есть коэффициент при них должен быть равен 1.
*/
Monomial normalisedMonomialLeastCommonMultiple(
    Monomial const& monomial1, Monomial const& monomial2)
{
    assert(monomial1.coefficient == 1);
    assert(monomial2.coefficient == 1);
    Monomial lcm = monomial1;
    for (size_t i = 0; i < monomial1.degrees.size(); ++i) {
        lcm.degrees[i] = std::max(monomial1.degrees[i], monomial2.degrees[i]);
    }
    return lcm;
}

/*
  Возвращает результат деления divisible на divisor.
  divisor должен быть нормализован,
  то есть коэффициент при этом одночлене должен быть равен 1.
*/
Monomial dividedByNormalisedMonomial(Monomial const& divisible, Monomial const& divisor)
{
    assert(divisor.coefficient == 1);
    auto res = divisible;
    for (size_t i = 0; i < divisible.degrees.size(); ++i) {
        res.degrees[i] -= divisor.degrees[i];
    }
    return res;
}

Polynomial multiplyByNormalisedMonomial(Polynomial const& polynomial, Monomial const& monomial)
{
    assert(monomial.coefficient == 1);
    auto res = polynomial;
    for (size_t monomialI = 0; monomialI < polynomial.monomials.size(); ++monomialI) {
        for (size_t i = 0; i < monomial.degrees.size(); ++i) {
            res.monomials[monomialI].degrees[i] += monomial.degrees[i];
        }
    }
    return res;
}

void sortPolynomial(Polynomial& polynomial)
{
    std::sort(polynomial.monomials.begin(), polynomial.monomials.end(), isMonomialGreater);
}

void sortPolynomialBasis(PolynomialBasis& basis)
{
    std::sort(basis.polynomials.begin(), basis.polynomials.end(), isPolynomialGreater);
}

void addMonomial(Polynomial& polynomial, Monomial const& monomial, int prime) {
    for (size_t i = 0; i < polynomial.monomials.size(); ++i) {
        if (polynomial.monomials[i].degrees == monomial.degrees) {
            polynomial.monomials[i].coefficient += monomial.coefficient;
            mod(polynomial.monomials[i].coefficient, prime);
            if (!polynomial.monomials[i].coefficient) {
                polynomial.monomials.erase(polynomial.monomials.begin() + i);
            }
            return;
        }
    }
    polynomial.monomials.push_back(monomial);
}

/*
  Возвращает результат умножения polynomial на -1.
*/
Polynomial negate(Polynomial const& polynomial, int prime)
{
    auto current = polynomial;
    for (auto& monomial : current.monomials) {
        monomial.coefficient = prime - monomial.coefficient;
    }
    return current;
}

Polynomial addPolynomials(Polynomial const& polynomial1, Polynomial const& polynomial2, int prime) {
    Polynomial result{ polynomial1.monomials };
    for (const auto& monomial : polynomial2.monomials) {
        addMonomial(result, monomial, prime);
    }
    sortPolynomial(result);
    return result;
}

void normalise(Polynomial& polynomial, int prime)
{
    assert(isSorted(polynomial));
    if (polynomial.monomials.size() > 0) {
        auto const major = getMajorMonomial(polynomial);
        int coefficient = getInverseElement(major.coefficient, prime);
        for (auto& monomial : polynomial.monomials) {
            monomial.coefficient *= coefficient;
            mod(monomial.coefficient, prime);
        }
    }
}

Polynomial generateRandomSortedPolynomial(size_t nVariables, int maxVariableDegree,
    int prime, size_t maxNMonomials)
{
    size_t nMonomials = rand() % (maxNMonomials + 1);
    Polynomial result;
    for (size_t i = 0; i < nMonomials; ++i) {
        std::vector<int> degrees;
        for (size_t variableI = 0; variableI < nVariables; ++variableI) {
            int degree = rand() % (maxVariableDegree + 1);
            degrees.push_back(degree);
        }
        int coefficient = 1 + rand() % (prime - 1);
        Monomial monomial{ degrees, coefficient };
        addMonomial(result, monomial, prime);
    }
    sortPolynomial(result);
    normalise(result, prime);
    return result;
}

void outputMonomial(Monomial const& monomial) {
    char letter = 'a';
    if (monomial.coefficient != 1) {
        std::cout << monomial.coefficient;
    }
    for (size_t i = 0; i < monomial.degrees.size(); ++i) {
        if (monomial.degrees[i]) {
            std::cout << letter;
            if (monomial.degrees[i] > 1) {
                std::cout << "^" << monomial.degrees[i];
            }
            if (i != monomial.degrees.size() - 1) {
                std::cout << "";
            }
        }
        ++letter;
    }
}

void outputPolynomial(Polynomial const& polynomial)
{
    for (size_t i = 0; i < polynomial.monomials.size(); ++i) {
        outputMonomial(polynomial.monomials[i]);
        if (i != polynomial.monomials.size() - 1) {
            std::cout << " + ";
        }
    }
    std::cout << std::endl;
}

void outputPolynomialBasis(PolynomialBasis const& basis)
{
    for (auto const& polynomial : basis.polynomials) {
        outputPolynomial(polynomial);
    }
    std::cout << std::endl;
}

void subtract(Polynomial& polynomial1, Polynomial const& polynomial2, int prime)
{
    polynomial1 = addPolynomials(polynomial1, negate(polynomial2, prime), prime);
}

Polynomial multipliedByCoefficient(Polynomial const& polynomial, int coefficient, int prime)
{
    auto res = polynomial;
    for (size_t monomialI = 0; monomialI < polynomial.monomials.size(); ++monomialI) {
        res.monomials[monomialI].coefficient *= coefficient;
        mod(res.monomials[monomialI].coefficient, prime);
        if (res.monomials[monomialI].coefficient == 0) {
            res.monomials.erase(res.monomials.begin() + monomialI);
        }
    }
    return res;
}

Polynomial multipliedByMonomial(Polynomial const& polynomial, Monomial const& monomial, int prime)
{
    auto res = polynomial;
    for (size_t monomialI = 0; monomialI < polynomial.monomials.size(); ++monomialI) {
        for (size_t i = 0; i < monomial.degrees.size(); ++i) {
            res.monomials[monomialI].degrees[i] += monomial.degrees[i];
        }
        res.monomials[monomialI].coefficient *= monomial.coefficient;
        mod(res.monomials[monomialI].coefficient, prime);
        if (res.monomials[monomialI].coefficient == 0) {
            res.monomials.erase(res.monomials.begin() + monomialI);
        }
    }
    return res;
}

bool isMonomialDivide(Monomial const& monomial1, Monomial const& monomial2)
{
    assert(monomial1.degrees.size() == monomial2.degrees.size());
    for (size_t i = 0; i < monomial1.degrees.size(); ++i) {
        if (monomial1.degrees[i] > monomial2.degrees[i]) {
            return false;
        }
    }
    return true;
}

Polynomial getReducedPolynomial(Polynomial const& polynomial,
    PolynomialBasis const& basis, int prime)
{
    assert(isSorted(basis));
    auto current = polynomial;
    for (size_t basisPolynomialI = 0;
        basisPolynomialI < basis.polynomials.size();
        ++basisPolynomialI) {
        auto basisPolynomial = basis.polynomials[basisPolynomialI];
        auto major = getMajorMonomial(basisPolynomial);
        for (size_t monomialI = 0; monomialI < current.monomials.size(); ++monomialI) {
            auto monomial = current.monomials[monomialI];
            if (isMonomialDivide(major, monomial)) {
                auto multiplier = dividedByNormalisedMonomial(monomial, major);
                auto multipliedBasisPolynomial =
                    multipliedByMonomial(basisPolynomial, multiplier, prime);
                subtract(current, multipliedBasisPolynomial, prime);
                monomialI = 0;
            }
        }
    }
    normalise(current, prime);
    return current;
}

Polynomial getPolynomialWithEliminatedMajorMonomials(
    Polynomial const& polynomial1, Polynomial const& polynomial2, int prime)
{
    auto major1 = getMajorMonomial(polynomial1);
    auto major2 = getMajorMonomial(polynomial2);
    auto lcm = normalisedMonomialLeastCommonMultiple(major1, major2);
    auto multiplier1 = dividedByNormalisedMonomial(lcm, major1);
    auto multiplier2 = dividedByNormalisedMonomial(lcm, major2);
    auto multipliedPolynomial1 = multipliedByMonomial(polynomial1, multiplier1, prime);
    auto multipliedPolynomial2 = multipliedByMonomial(polynomial2, multiplier2, prime);
    subtract(multipliedPolynomial1, multipliedPolynomial2, prime);
    return multipliedPolynomial1;
}

Polynomial getReducedSPolynomial(PolynomialBasis const& basis,
    Polynomial const& polynomial1, Polynomial const& polynomial2, int prime)
{
    auto eliminated = getPolynomialWithEliminatedMajorMonomials(
        polynomial1, polynomial2, prime);
    auto reduced = getReducedPolynomial(eliminated, basis, prime);
    return reduced;
}

bool isZero(Polynomial const& polynomial)
{
    return polynomial.monomials.size() == 0;
}

Polynomial getFirstNotZeroSPolynomial(PolynomialBasis const& basis, int prime)
{
    for (auto const& a : basis.polynomials) {
        for (auto const& b : basis.polynomials) {
            auto sPolynomial = getReducedSPolynomial(basis, a, b, prime);
            if (sPolynomial.monomials.size() != 0) {
                return sPolynomial;
            }
        }
    }
    return { {} };
}

PolynomialBasis getGrobnerBasis(PolynomialBasis const& initialBasis, int prime)
{
    auto currentBasis = initialBasis;
    sortPolynomialBasis(currentBasis);
    while (true) {
        outputPolynomialBasis(currentBasis);
        auto s = getFirstNotZeroSPolynomial(currentBasis, prime);
        if (isZero(s)) {
            break;
        }
        else {
            currentBasis.polynomials.push_back(s);
            sortPolynomialBasis(currentBasis);
        }
    }
    return currentBasis;
}

PolynomialBasis generateRandomPolynomialBasis(size_t nVariables, int maxVariableDegree,
    int prime, size_t maxNMonomials, size_t nPolynomials)
{
    PolynomialBasis currentBasis;
    while (currentBasis.polynomials.size() < nPolynomials) {
        auto newPolynomial = generateRandomSortedPolynomial(
            nVariables, maxVariableDegree, prime, maxNMonomials);
        if (!isZero(newPolynomial)) {
            currentBasis.polynomials.push_back(newPolynomial);
        }
    }
    return currentBasis;
}

/*
    Возвращает случайный ненулевой многочлен из идеала, создаваемого базисом basis.
*/
Polynomial generateRandomPolynomialFromIdeal(PolynomialBasis const& basis, int prime)
{
    Polynomial current;
    for (auto const& polynomial : basis.polynomials) {
        auto randomCoefficient = rand() % prime;
        auto multiplied = multipliedByCoefficient(polynomial, randomCoefficient, prime);
        subtract(current, multiplied, prime);
    }
    return isZero(current) ?
        generateRandomPolynomialFromIdeal(basis, prime) :
        current;
}

/*
    Определяет, выполняется ли свойство базиса Грёбнера для отдельного
    многочлена из идеала
*/
bool isGrobnerForPolynomial(PolynomialBasis const& grobnerBasis,
    Polynomial const& polynomialFromIdeal)
{
    for (auto const& grobnerBasisPolynomial : grobnerBasis.polynomials) {
        auto idealMajor = getMajorMonomial(polynomialFromIdeal);
        auto grobnerMajor = getMajorMonomial(grobnerBasisPolynomial);
        if (isMonomialDivide(grobnerMajor, idealMajor)) {
            return true;
        }
    }
    return false;
}

bool isGrobnerBasis(PolynomialBasis const& initialBasis,
    PolynomialBasis const& grobnerBasis, int prime)
{
    for (int i = 0; i < 100; ++i) {
        auto polynomialFromIdeal = generateRandomPolynomialFromIdeal(initialBasis, prime);
        if (!isGrobnerForPolynomial(grobnerBasis, polynomialFromIdeal)) {
            return false;
        }
    }
    return true;
}

void testNegate()
{
    auto poly1 = Polynomial{ {} };
    assert(polynomialEqual(negate(poly1, 7), poly1));
    auto poly2 = Polynomial("a^2*b^3 + 4*a^2*b^2", 2);
    auto poly3 = Polynomial("6*a^2*b^3 + 3*a^2*b^2", 2);
    assert(polynomialEqual(negate(poly2, 7), poly3));
}

void testIsMonomialLess()
{
    auto monomial1 = Monomial("a^2*b^3", 2);
    auto monomial2 = Monomial("a^2*b^2", 2);
    assert(isMonomialGreater(monomial1, monomial2));
    assert(!isMonomialGreater(monomial2, monomial1));
    auto monomial3 = Monomial("a^2*b^3", 2);
    auto monomial4 = Monomial("a^1*b^5", 2);
    assert(isMonomialGreater(monomial3, monomial4));
    assert(!isMonomialGreater(monomial4, monomial3));
}

void testIsSorted()
{
    auto polynomial1 = Polynomial("a^1*b^5 + 2*a^2*b^4", 2);
    assert(!isSorted(polynomial1));
    auto polynomial2 = Polynomial("a^3*b^2 + 2*a^2*b^4", 2);
    assert(isSorted(polynomial2));
}

void testNormalisedMonomialDivide()
{
    auto divisible1 = Monomial("2*a^2*b^3", 2);
    auto divisor1 = Monomial("1", 2);
    auto expected1 = Monomial("2*a^2*b^3", 2);
    auto result1 = dividedByNormalisedMonomial(divisible1, divisor1);

    assert(result1 == expected1);
    auto divisor2 = Monomial("a^2*b^3", 2);
    auto expected2 = Monomial("2", 2);
    auto result2 = dividedByNormalisedMonomial(divisible1, divisor2);
    assert(result2 == expected2);

    auto divisor3 = Monomial("a^1*b^1", 2);
    auto expected3 = Monomial("2*a^1*b^2", 2);
    auto result3 = dividedByNormalisedMonomial(divisible1, divisor3);
    assert(result3 == expected3);
}

void testMultiplyByNormalisedMonomial()
{
    auto polynomial1 = Polynomial("a^1*b^4", 2);
    auto monomial1 = Monomial("a^1*b^1", 2);
    auto expected1 = Polynomial("a^2*b^5", 2);
    auto result1 = multiplyByNormalisedMonomial(polynomial1, monomial1);
    assert(polynomialEqual(result1, expected1));
    auto polynomial2 = Polynomial("a^3*b^4 + a^1*b^2", 2);
    auto expected2 = Polynomial("a^4*b^5 + a^2*b^3", 2);
    auto result2 = multiplyByNormalisedMonomial(polynomial2, monomial1);
    assert(polynomialEqual(result2, expected2));
    auto polynomial3 = Polynomial("1", 2);
    auto monomial3 = Monomial("1", 2);
    auto expected3 = Polynomial("1", 2);
    auto result3 = multiplyByNormalisedMonomial(polynomial3, monomial3);
    assert(polynomialEqual(result3, expected3));
    auto polynomial4 = Polynomial{ {} };
    auto monomial4 = Monomial("a^5", 2);
    auto expected4 = Polynomial{ {} };
    auto result4 = multiplyByNormalisedMonomial(polynomial4, monomial4);
    assert(polynomialEqual(result4, expected4));
}

void testGenerateRandomSortedPolynomial()
{
    for (size_t i = 0; i < 20; ++i)
    {
        auto poly = generateRandomSortedPolynomial(3, 5, 19, 5);
        outputPolynomial(poly);
    }
    for (size_t i = 0; i < 100; ++i)
    {
        auto poly = generateRandomSortedPolynomial(32, 50, 19, 50);
        assert(isSorted(poly));
    }
}

void testSubtract()
{
    auto poly1 = Polynomial();
    subtract(poly1, { {} }, 7);
    assert(polynomialEqual(poly1, { {} }));

    auto poly2 = Polynomial();
    subtract(poly2, { "5*a^2*b^3", 2 }, 7);
    assert(polynomialEqual(poly2, { "2*a^2*b^3", 2 }));

    auto poly3 = Polynomial("5*a^2*b^3", 2);
    subtract(poly3, { {} }, 7);
    assert(polynomialEqual(poly3, { "5*a^2*b^3", 2 }));

    auto poly4 = Polynomial("5*a^2*b^3", 2);
    subtract(poly4, { "2*a^2*b^3", 2 }, 7);
    assert(polynomialEqual(poly4, { "3*a^2*b^3", 2 }));

    auto poly5 = Polynomial("5*a^2*b^3", 2);
    subtract(poly5, { "5*a^2*b^3", 2 }, 7);
    assert(polynomialEqual(poly5, { {} }));

    auto poly6 = Polynomial("2*a^2*b^3", 2);
    subtract(poly6, { "3*a^2*b^3", 2 }, 7);
    assert(polynomialEqual(poly6, { "6*a^2*b^3", 2 }));

    auto poly7 = Polynomial("a^3*b^3", 2);
    subtract(poly7, { "a^2*b^2", 2 }, 5);
    assert(polynomialEqual(poly7, { "a^3*b^3 + 4*a^2*b^2", 2 }));

    auto poly8 = Polynomial("a^2*b^2", 2);
    subtract(poly8, { "a^3*b^3", 2 }, 5);
    assert(polynomialEqual(poly8, { "4*a^3*b^3 + a^2*b^2", 2 }));

    auto poly9 = Polynomial("2*a^4*b^4 + 3*a^2*b^2", 2);
    subtract(poly9, { "4*a^3*b^3 + 3*a^1*b^1", 2 }, 5);
    assert(polynomialEqual(poly9, { "2*a^4*b^4 + a^3*b^3 + 3*a^2*b^2 + 2*a^1*b^1", 2 }));

    auto poly10 = Polynomial("a^1*b^2", 2);
    subtract(poly10, { "a^2*b^1", 2 }, 5);
    assert(polynomialEqual(poly10, { "4*a^2*b^1 + a^1*b^2", 2 }));
}

void testMultipliedByMonomial()
{
    auto poly0 = Polynomial("1", 2);
    auto monomial0 = Monomial("1", 2);
    auto expected0 = Polynomial("1", 2);
    auto result0 = multipliedByMonomial(poly0, monomial0, 5);
    assert(polynomialEqual(expected0, result0));

    auto poly1 = Polynomial("a^1*b^4", 2);
    auto monomial1 = Monomial("a^1*b^1", 2);
    auto expected1 = Polynomial("a^2*b^5", 2);
    auto result1 = multipliedByMonomial(poly1, monomial1, 5);
    assert(polynomialEqual(expected1, result1));

    auto poly2 = Polynomial("a^3*b^4 + a^1*b^2", 2);
    auto monomial2 = Monomial("a^1*b^1", 2);
    auto expected2 = Polynomial("a^4*b^5 + a^2*b^3", 2);
    auto result2 = multipliedByMonomial(poly2, monomial2, 5);
    assert(polynomialEqual(expected2, result2));

    auto poly3 = Polynomial("a^1*b^1", 2);
    auto monomial3 = Monomial("2", 2);
    auto expected3 = Polynomial("2*a^1*b^1", 2);
    auto result3 = multipliedByMonomial(poly3, monomial3, 5);
    assert(polynomialEqual(expected3, result3));
}

void testGetReducedPolynomial()
{
    auto poly0 = Polynomial();
    auto poly1 = Polynomial("1", 1);
    auto result1 = getReducedPolynomial(poly1, { { poly1 } }, 7);
    assert(polynomialEqual(result1, poly0));

    auto poly2 = Polynomial("a^1", 1);
    auto result2 = getReducedPolynomial(poly2, { { poly1 } }, 7);
    assert(polynomialEqual(result2, poly0));

    auto poly3 = Polynomial("a^2", 1);
    auto poly4 = Polynomial("a^5", 1);
    auto result3 = getReducedPolynomial(poly4, { { poly3 } }, 7);
    assert(polynomialEqual(result3, poly0));

    auto result4 = getReducedPolynomial(poly3, { { poly4 } }, 7);
    assert(polynomialEqual(result4, poly3));

    auto poly5 = Polynomial("a^1 + b^1", 2);
    auto poly6 = Polynomial("a^1", 2);
    auto poly7 = Polynomial("b^1", 2);
    auto result5 = getReducedPolynomial(poly6, { { poly5 } }, 7);
    assert(polynomialEqual(result5, poly7));

    auto poly8 = Polynomial("b^1 + c^1", 3);
    auto poly9 = Polynomial("a^1 + b^1", 3);
    auto poly10 = Polynomial("a^1", 3);
    auto poly11 = Polynomial("c^1", 3);
    auto basis0 = PolynomialBasis{ { poly8, poly9 } };
    sortPolynomialBasis(basis0);
    auto result6 = getReducedPolynomial(poly10, basis0, 7);
    assert(polynomialEqual(result6, poly11));

    auto result7 = getReducedPolynomial(poly8, basis0, 7);
    assert(polynomialEqual(result7, { {} }));

    auto poly12 = Polynomial("3*a^1", 2);
    auto poly13 = Polynomial("b^1", 2);
    auto result8 = getReducedPolynomial(poly12, { { poly5 } }, 7);
    assert(polynomialEqual(result8, poly13));
}

void testGetPolynomialWithEliminatedMajorMonomials()
{
    auto result1 = getPolynomialWithEliminatedMajorMonomials(
        { "1", 1 }, { "1", 1 }, 7);
    assert(polynomialEqual(result1, { {} }));

    auto result2 = getPolynomialWithEliminatedMajorMonomials(
        { "a^1", 2 }, { "b^1", 2 }, 7);
    assert(polynomialEqual(result2, { {} }));

    auto result3 = getPolynomialWithEliminatedMajorMonomials(
        { "a^1 + b^1", 2 }, { "a^1", 2 }, 7);
    assert(polynomialEqual(result3, { "b^1", 2 }));

    auto result4 = getPolynomialWithEliminatedMajorMonomials(
        { "a^1", 2 }, { "a^1 + b^1", 2 }, 7);
    assert(polynomialEqual(result4, { "6*b^1", 2 }));
}

void testGetGrobnerBasis(size_t nVariables, size_t nPolynomials, int maxVariableDegree,
    size_t maxNMonomials, int prime, int nIterations)
{
    for (int i = 0; i < nIterations; ++i) {
        auto basis = generateRandomPolynomialBasis(
            nVariables, maxVariableDegree, prime, maxNMonomials, nPolynomials);
        auto grobnerBasis = getGrobnerBasis(basis, prime);
        assert(isGrobnerBasis(basis, grobnerBasis, prime));
    }
}

void testSortPolynomialBasis()
{
    auto const poly1 = Polynomial("a^1 + 2*b^2", 2);
    auto const poly2 = Polynomial("a^1 + b^6", 2);
    auto basis = PolynomialBasis{ { poly1, poly2 } };
    sortPolynomialBasis(basis);
    auto expected = PolynomialBasis{ { poly2, poly1 } };
    assert(basisEqual(basis, expected));
}

void testGetFirstNotZeroSPolynomial_1()
{
    auto poly1 = Polynomial("a^2*b^2 + a^1", 2);
    auto poly2 = Polynomial("a^2*b^1 + b^1", 2);
    auto basis = PolynomialBasis{ { poly1, poly2 } };
    auto result = getFirstNotZeroSPolynomial(basis, 3);
    auto expected = Polynomial("a^1 + 2*b^2", 2);
    assert(polynomialEqual(result, expected));
}

void testGetGrobnerBasis()
{
    srand(0);
    //nVariables, nPolynomials, maxVariableDegree, maxNMonomials, prime, nIterations
    //testGetGrobnerBasis(1, 1, 1, 1, 2, 10);
    testGetGrobnerBasis(2, 2, 2, 2, 3, 100);
}

void testAll()
{
    testNegate();
    testIsMonomialLess();
    testIsSorted();
    testNormalisedMonomialDivide();
    testMultiplyByNormalisedMonomial();
    testGenerateRandomSortedPolynomial();
    testSubtract();
    testMultipliedByMonomial();
    testGetReducedPolynomial();
    testGetPolynomialWithEliminatedMajorMonomials();
    testSortPolynomialBasis();
    testGetFirstNotZeroSPolynomial_1();
    // testGetGrobnerBasis();
}

int main(void)
{
    testAll();
    return 0;
}